#include "hip/hip_runtime.h"
#include "CudaPlotContext.h"

//-----------------------------------------------------------
__global__ void GenSortKey( const uint32 entryCount, uint32* key )
{
    const uint32 gid = blockIdx.x * blockDim.x + threadIdx.x;
    if( gid >= entryCount )
        return;

    key[gid] = gid;
}

//-----------------------------------------------------------
void CudaK32PlotGenSortKey( const uint32 entryCount, uint32* devKey, hipStream_t stream, bool synchronize )
{
    const uint32 threadsPerBlock = 128;
    const uint32 blockCount      = CDiv( entryCount, threadsPerBlock );
    
    if( stream == nullptr )
        stream = hipStreamLegacy;

    GenSortKey<<<blockCount, threadsPerBlock, 0, stream>>>( entryCount, devKey );
    if( synchronize )
        CudaErrCheck( hipStreamSynchronize( stream ) );
    
}

//-----------------------------------------------------------
template<typename T>
__global__ void SortByKey( const uint32 entryCount, const uint32* key, const T* input, T* output )
{
    const uint32 gid = blockIdx.x * blockDim.x + threadIdx.x;
    if( gid >= entryCount )
        return;

    output[gid] = input[key[gid]];
}


//-----------------------------------------------------------
template<typename T>
void CudaK32PlotSortByKey( const uint32 entryCount, const uint32* devKey, const T* devInput, T* devOutput, hipStream_t stream, bool synchronize )
{
    const uint32 threadsPerBlock = 128;
    const uint32 blockCount      = CDiv( entryCount, threadsPerBlock );
    
    if( stream == nullptr )
        stream = hipStreamLegacy;

    SortByKey<T><<<blockCount, threadsPerBlock, 0, stream>>>( entryCount, devKey, devInput, devOutput );
    if( synchronize )
        CudaErrCheck( hipStreamSynchronize( stream ) );
}

//-----------------------------------------------------------
void CudaK32PlotSortMeta( const uint32 entryCount, const uint32* devKey, const uint32* devMetaIn, uint32* devMetaOutput, hipStream_t stream )
{

}


template void CudaK32PlotSortByKey<uint16>( const uint32 entryCount, const uint32* devKey, const uint16* devInput, uint16* devOutput, hipStream_t stream, bool synchronize );
template void CudaK32PlotSortByKey<uint32>( const uint32 entryCount, const uint32* devKey, const uint32* devInput, uint32* devOutput, hipStream_t stream, bool synchronize );
template void CudaK32PlotSortByKey<uint64>( const uint32 entryCount, const uint32* devKey, const uint64* devInput, uint64* devOutput, hipStream_t stream, bool synchronize );
template void CudaK32PlotSortByKey<K32Meta3>( const uint32 entryCount, const uint32* devKey, const K32Meta3* devInput, K32Meta3* devOutput, hipStream_t stream, bool synchronize );
template void CudaK32PlotSortByKey<K32Meta4>( const uint32 entryCount, const uint32* devKey, const K32Meta4* devInput, K32Meta4* devOutput, hipStream_t stream, bool synchronize );
template void CudaK32PlotSortByKey<Pair>( const uint32 entryCount, const uint32* devKey, const Pair* devInput, Pair* devOutput, hipStream_t stream, bool synchronize );

